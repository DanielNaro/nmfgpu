#include "hip/hip_runtime.h"
#include <common/Logging.h>
#include <common/MatrixDescription.h>
#include <nmf/KernelHelper.cuh>
#include <nmf/KernelMakeNonNegative.h>

namespace nmfgpu {
	namespace kernel {
		namespace impl {
			template<typename NumericType, bool transpose>
			__global__ void kernelMakeNonNegative(unsigned m, unsigned n, const NumericType* A, unsigned ldda, NumericType* B, unsigned lddb) {
				unsigned column = blockIdx.x * blockDim.x + threadIdx.x;
				unsigned row = blockIdx.y * blockDim.y + threadIdx.y;

				if (row >= m || column >= n)
					return;

				NumericType value;
				if (transpose) {
					value = CUDA_READ_ONLY_CACHE(A[row * ldda + column]);
				} else {
					value = A[column * ldda + row];
				}

				B[column * lddb + row] = max(value, NumericType(0.f));
			}
		}

		void makeNonNegative(const MatrixDescription<float>& matrix, hipStream_t stream) {
			if (matrix.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(matrix.rows);
			auto n = static_cast<unsigned>(matrix.columns);
			auto ld = static_cast<unsigned>(matrix.dense.leadingDimension);
			dim3 blockDim(32, 32);
			dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);
			impl::kernelMakeNonNegative<float, false><<<gridDim, blockDim, 0, stream>>>(m, n, matrix.dense.values, ld, matrix.dense.values, ld);
		}

		void makeNonNegative(const MatrixDescription<double>& matrix, hipStream_t stream) {
			if (matrix.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(matrix.rows);
			auto n = static_cast<unsigned>(matrix.columns);
			auto ld = static_cast<unsigned>(matrix.dense.leadingDimension);
			dim3 blockDim(32, 32);
			dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);
			impl::kernelMakeNonNegative<double, false><<<gridDim, blockDim, 0, stream>>>(m, n, matrix.dense.values, ld, matrix.dense.values, ld);
		}
	}
}
