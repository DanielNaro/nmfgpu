#include "hip/hip_runtime.h"
#include <common/Logging.h>
#include <common/MatrixDescription.h>
#include <nmf/KernelNormalizeColumns.h>

namespace nmfgpu {
	namespace kernel {
		namespace impl {
			template<typename NumericType>
			__global__ void normalizeColumns(unsigned m, unsigned n, NumericType* A, unsigned ldda) {
				unsigned column = blockDim.y * blockIdx.y + threadIdx.y;

				if (column >= n) {
					return;
				}

				// Compute sum of column
				NumericType sum = 0;

				for (unsigned i = threadIdx.x; i < m; i += blockDim.x) {
					sum += A[column * ldda + i] * A[column * ldda + i];
				}

				// Using warp reduction to get column sum
				sum += __shfl_xor(sum, 16);
				sum += __shfl_xor(sum, 8);
				sum += __shfl_xor(sum, 4);
				sum += __shfl_xor(sum, 2);
				sum += __shfl_xor(sum, 1);

				// Normalize column
				if (sum > 0) {
					sum = sqrt(sum);
					for (unsigned i = threadIdx.x; i < m; i += blockDim.x) {
						NumericType value = A[column * ldda + i];
						A[column * ldda + i] = value / sum;
					}
				}
			}
		}

		void normalizeColumns(const MatrixDescription<float>& target, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 8);
			dim3 gridDim(1, (n + blockDim.y - 1) / blockDim.y);
			impl::normalizeColumns<float><<<gridDim, blockDim, 0, stream>>>(m, n, target.dense.values, static_cast<unsigned>(target.dense.leadingDimension));
		}

		void normalizeColumns(const MatrixDescription<double>& target, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 8);
			dim3 gridDim(1, (n + blockDim.y - 1) / blockDim.y);
			impl::normalizeColumns<double><<<gridDim, blockDim, 0, stream>>>(m, n, target.dense.values, static_cast<unsigned>(target.dense.leadingDimension));
		}
	}
}
