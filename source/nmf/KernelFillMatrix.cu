#include "hip/hip_runtime.h"
/*
nmfgpu - CUDA accelerated computation of Non-negative Matrix Factorizations (NMF)

Copyright (C) 2015-2016  Sven Koitka (sven.koitka@fh-dortmund.de)

This file is part of nmfgpu.

nmfgpu is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

nmfgpu is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with nmfgpu.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <common/Logging.h>
#include <common/MatrixDescription.h>
#include <nmf/KernelMultiplyDivide.h>

namespace nmfgpu {
	namespace kernel {
		namespace impl {
			template<typename NumericType, bool ReuseValue>
			__global__ void fillMatrix(unsigned m, unsigned n, NumericType* A, unsigned lda, NumericType diagValue, NumericType offDiagValue) {
				unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
				unsigned column = blockIdx.y * blockDim.y + threadIdx.y;
				
				if(row >= m || column >= n)
					return;
				
				unsigned index = column * lda + row;
				
				const NumericType ReuseValueFactor = ReuseValue ? 1.0 : 0.0;
				if (row == column) {
					A[index] = A[index] * ReuseValueFactor + diagValue;
				} else {
					A[index] = A[index] * ReuseValueFactor + offDiagValue;
				}
			}
		}

		void fillMatrix(const MatrixDescription<float>& target, float diagValue, float offDiagValue, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 32);
			dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
			impl::fillMatrix<float, false><<<gridDim, blockDim, 0, stream>>>(m, n, target.dense.values, static_cast<unsigned>(target.dense.leadingDimension), diagValue, offDiagValue);
		}

		void fillMatrix(const MatrixDescription<double>& target, double diagValue, double offDiagValue, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 32);
			dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
			impl::fillMatrix<double, false><<<gridDim, blockDim, 0, stream>>>(m, n, target.dense.values, static_cast<unsigned>(target.dense.leadingDimension), diagValue, offDiagValue);
		}

		void addConstantToMatrix(const MatrixDescription<float>& target, float diagValue, float offDiagValue, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 32);
			dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
			impl::fillMatrix<float, true><<<gridDim, blockDim, 0, stream >> >(m, n, target.dense.values, static_cast<unsigned>(target.dense.leadingDimension), diagValue, offDiagValue);
		}

		void addConstantToMatrix(const MatrixDescription<double>& target, double diagValue, double offDiagValue, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrix must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 32);
			dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
			impl::fillMatrix<double, true><<<gridDim, blockDim, 0, stream >> >(m, n, target.dense.values, static_cast<unsigned>(target.dense.leadingDimension), diagValue, offDiagValue);
		}
	}
}
