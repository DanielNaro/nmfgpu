#include "hip/hip_runtime.h"
/*
nmfgpu - CUDA accelerated computation of Non-negative Matrix Factorizations (NMF)

Copyright (C) 2015-2016  Sven Koitka (sven.koitka@fh-dortmund.de)

This file is part of nmfgpu.

nmfgpu is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

nmfgpu is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with nmfgpu.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <common/Logging.h>
#include <common/MatrixDescription.h>
#include <nmf/KernelMultiplyDivide.h>

namespace nmfgpu {
	namespace kernel {
		namespace impl {
			template<typename NumericType>
			__global__ void multiplyDivide(unsigned m, unsigned n, NumericType* source, const NumericType* numerator, const NumericType* denominator, unsigned lddabc, NumericType epsilon) {
				unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
				unsigned column = blockIdx.y * blockDim.y + threadIdx.y;
				
				if(row >= m || column >= n)
					return;
				
				unsigned index = column * lddabc + row;
				
				NumericType value = source[index];
				NumericType upper = numerator[index];
				NumericType lower = denominator[index];
				source[index] = value * upper / (lower + epsilon);
			}
		}

		void multiplyDivide(const MatrixDescription<float>& target, const MatrixDescription<float>& numerator, const MatrixDescription<float>& denominator, float epsilon, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense || numerator.format != StorageFormat::Dense || denominator.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrices must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 32);
			dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
			impl::multiplyDivide<float><<<gridDim, blockDim, 0, stream>>>(m, n, target.dense.values, numerator.dense.values, denominator.dense.values, denominator.dense.leadingDimension, epsilon);
		}

		void multiplyDivide(const MatrixDescription<double>& target, const MatrixDescription<double>& numerator, const MatrixDescription<double>& denominator, double epsilon, hipStream_t stream /* = nullptr */) {
			if (target.format != StorageFormat::Dense || numerator.format != StorageFormat::Dense || denominator.format != StorageFormat::Dense) {
				Logging::instance().error()
					.print("[ERROR] " NMFGPU_FILE_LINE_PREFIX ": Input matrices must be stored in dense format!").lineFeed();
				return;
			}

			auto m = static_cast<unsigned>(target.rows);
			auto n = static_cast<unsigned>(target.columns);
			dim3 blockDim(32, 32);
			dim3 gridDim((m + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);
			impl::multiplyDivide<double><<<gridDim, blockDim, 0, stream>>>(m, n, target.dense.values, numerator.dense.values, denominator.dense.values, denominator.dense.leadingDimension, epsilon);
		}
	}
}
